#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""

#include <iostream>
#include <fstream>
#include <math.h>
#include <float.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#include "Utility.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include "metal.h"
#include "lambertian.h"
#include "dielectric.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray &r, hitable ** world, hiprandState * local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);

    for (int i = 0; i < 100; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0, 0, 0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5 * (unit_direction.y() + 1.0); // 0 <= t <= 1
            // lerp between white and blue
            // lerp := (1-t) * start_value + t * end_value;
            vec3 c = (1.0 - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0);
}

__global__ void rand_init(hiprandState * rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState * rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    // Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// __global__ called from host, executed on device
__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera ** cam, hitable ** world, hiprandState * rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new sphere(vec3(0, -1000.0, -1), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
        
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8) {
                    d_list[i++] = new sphere(center, 0.2,
                        new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
                }
                else if (choose_mat < 0.95) {
                    d_list[i++] = new sphere(center, 0.2,
                        new metal(vec3(0.5*(1.0 + RND), 0.5*(1.0 + RND), 0.5*(1.0 + RND)), 0.5*RND));
                }
                else {
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
        d_list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new hitable_list(d_list, 22 * 22 + 1 + 3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.1;
        *d_camera = new camera(lookfrom, lookat, vec3(0, 1, 0), 30.0, float(nx) / float(ny), aperture, dist_to_focus);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

int main()
{

    std::ofstream outputFile;
    outputFile.open("cuda_canvas.ppm");

    int nx = 1200;
    int ny = 800;
    int ns = 100;

    int tx = 32;
    int ty = 32;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << nx / tx + 1 << "x" << ny / ty + 1 << " blocks with " << tx << "x" << ty << " threads per blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate fb (frame buffer)
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

    // 2nd random state is initialized for the world creation
    //rand_init<<<1,1>>>(d_rand_state2);
    rand_init<<<1, 1>>> (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make the world of hitables and the camera
    hitable **d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **)&d_list, num_hitables * sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world <<<1, 1>>> (d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();

    // render the buffer
    dim3 blocks(nx / tx + 2, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init <<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render <<<blocks, threads>>>(fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    outputFile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            outputFile << ir << " " << ig << " " << ib << "\n";
        }
    }
    float R = cos(M_PI / 4);

    checkCudaErrors(hipDeviceSynchronize());
    free_world <<<1, 1>>> (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();

    outputFile.close();
    //delete[] list;
    //delete world;
    return 0;
}

